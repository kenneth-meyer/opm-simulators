#include "hip/hip_runtime.h"
/*
  Copyright 2019 Equinor ASA

  This file is part of the Open Porous Media project (OPM).

  OPM is free software: you can redistribute it and/or modify
  it under the terms of the GNU General Public License as published by
  the Free Software Foundation, either version 3 of the License, or
  (at your option) any later version.

  OPM is distributed in the hope that it will be useful,
  but WITHOUT ANY WARRANTY; without even the implied warranty of
  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
  GNU General Public License for more details.

  You should have received a copy of the GNU General Public License
  along with OPM.  If not, see <http://www.gnu.org/licenses/>.
*/

#ifndef __NVCC__
    #error "Cannot compile for cusparse: NVIDIA compiler not found"
#endif

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#include <sys/time.h>
#include <sstream>
//include statement to write to csv file
#include <fstream>

#include <opm/common/OpmLog/OpmLog.hpp>

#include <opm/simulators/linalg/bda/cusparseSolverBackend.hpp>
#include <opm/simulators/linalg/bda/BdaResult.hpp>
#include <opm/simulators/linalg/bda/cuda_header.hpp>

#include "hipblas.h"
#include "hipsparse.h"
// For more information about cusparse, check https://docs.nvidia.com/cuda/cusparse/index.html

// iff true, the nonzeroes of the matrix are copied row-by-row into a contiguous, pinned memory array, then a single GPU memcpy is done
// otherwise, the nonzeroes of the matrix are assumed to be in a contiguous array, and a single GPU memcpy is enough
#define COPY_ROW_BY_ROW 0

namespace Opm
{

    const hipsparseSolvePolicy_t policy = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
    const hipsparseOperation_t operation  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    const hipsparseDirection_t order = HIPSPARSE_DIRECTION_ROW;

    double second(void) {
        struct timeval tv;
        gettimeofday(&tv, nullptr);
        return (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;
    }

    cusparseSolverBackend::cusparseSolverBackend(int verbosity_, int maxit_, double tolerance_) : verbosity(verbosity_), maxit(maxit_), tolerance(tolerance_), minit(0) {
    }

    cusparseSolverBackend::~cusparseSolverBackend() {
        finalize();
    }

    void cusparseSolverBackend::gpu_pbicgstab(WellContributions& wellContribs, BdaResult& res) {
        //added declaration of additional variables to keep time
        double t_total1, t_total2, t_wellContribs1, t_wellContribs2, t_matVecMult1, t_matVecMult2, t_triSolve1, t_triSolve2;
        double t_wellContribs_total = 0.0;
        double t_matVecMult_total = 0.0;
        double t_triSolve_total = 0.0;
        int n = N;
        double rho = 1.0, rhop;
        double alpha, nalpha, beta;
        double omega, nomega, tmp1, tmp2;
        double norm, norm_0;
        double zero = 0.0;
        double one  = 1.0;
        double mone = -1.0;
        float it;

        t_total1 = second();

        /* ----------------------------------------------------------------
         * TIMES FOR INDIVIDUAL STEPS WITHIN THE LINEAR_SOLVE_TIME CUMULATIVE TIME
         -----------------------------------------------------------------*/

        if(wellContribs.getNumWells() > 0){
            // START TIME
            t_wellContribs1 = second();
            wellContribs.setCudaStream(stream);
            // END TIME
            t_wellContribs2 = second();
            t_wellContribs_total += t_wellcontribs2 - t_wellContribs1;
        }

        hipsparseDbsrmv(cusparseHandle, order, operation, Nb, Nb, nnzb, &one, descr_M, d_bVals, d_bRows, d_bCols, block_size, d_x, &zero, d_r);

        hipblasDscal(cublasHandle, n, &mone, d_r, 1);
        hipblasDaxpy(cublasHandle, n, &one, d_b, 1, d_r, 1);
        hipblasDcopy(cublasHandle, n, d_r, 1, d_rw, 1);
        hipblasDcopy(cublasHandle, n, d_r, 1, d_p, 1); 
        hipblasDnrm2(cublasHandle, n, d_r, 1, &norm_0);

        if (verbosity > 1) {
            std::ostringstream out;
            out << std::scientific << "cusparseSolver initial norm: " << norm_0;
            OpmLog::info(out.str());
        }

        //loop where linear iterations occur
        for (it = 0.5; it < maxit; it+=0.5) {
            rhop = rho;
            hipblasDdot(cublasHandle, n, d_rw, 1, d_r, 1, &rho);

            if (it > 1) {
                beta = (rho/rhop) * (alpha/omega);
                nomega = -omega;
                hipblasDaxpy(cublasHandle, n, &nomega, d_v, 1, d_p, 1);
                hipblasDscal(cublasHandle, n, &beta, d_p, 1);
                hipblasDaxpy(cublasHandle, n, &one, d_r, 1, d_p, 1);
            }

            // THIS IS THE SPARSE TRIANGULAR MATRIX SOLVE PORTION OF THE ITERATION
            // START TIME
            t_triSolve1 = second();
            // apply ilu0
            hipsparseDbsrsv2_solve(cusparseHandle, order, \
                operation, Nb, nnzb, &one, \
                descr_L, d_mVals, d_mRows, d_mCols, block_size, info_L, d_p, d_t, policy, d_buffer);
            hipsparseDbsrsv2_solve(cusparseHandle, order, \
                operation, Nb, nnzb, &one, \
                descr_U, d_mVals, d_mRows, d_mCols, block_size, info_U, d_t, d_pw, policy, d_buffer);
            // END TIME
            t_triSolve2 = second();
            t_triSolve_total += t_triSolve2 - t_triSolve1;

            // SPARE MATRIX VECTOR MULTIPLICATION PORTION OF ITERATION
            // START TIME
            t_matVecMult1 = second();
            // spmv
            hipsparseDbsrmv(cusparseHandle, order, \
                operation, Nb, Nb, nnzb, \
                &one, descr_M, d_bVals, d_bRows, d_bCols, block_size, d_pw, &zero, d_v);
            // END TIME
            t_matVecMult2 = second();
            t_matVecMult_total += t_matVecMult2 - t_matVecMult1;

            // apply wellContributions
            if(wellContribs.getNumWells() > 0){
                // START TIME
                t_wellContribs1 = second();
                wellContribs.apply(d_pw, d_v);
                // END TIME
                t_wellContribs2 = second();
                t_wellContribs_total += t_wellcontribs2 - t_wellContribs1;
            }

            hipblasDdot(cublasHandle, n, d_rw, 1, d_v, 1, &tmp1);
            alpha = rho / tmp1;
            nalpha = -alpha;
            hipblasDaxpy(cublasHandle, n, &nalpha, d_v, 1, d_r, 1);
            hipblasDaxpy(cublasHandle, n, &alpha, d_pw, 1, d_x, 1);
            hipblasDnrm2(cublasHandle, n, d_r, 1, &norm);

            if (norm < tolerance * norm_0 && it > minit) {
                break;
            }

            it += 0.5;

            // THIS IS THE SPARSE TRIANGULAR MATRIX SOLVE PORTION OF THE ITERATION (second time around)
            // START TIME
            t_triSolve1 = second();
            // apply ilu0
            hipsparseDbsrsv2_solve(cusparseHandle, order, \
                operation, Nb, nnzb, &one, \
                descr_L, d_mVals, d_mRows, d_mCols, block_size, info_L, d_r, d_t, policy, d_buffer);
            hipsparseDbsrsv2_solve(cusparseHandle, order, \
                operation, Nb, nnzb, &one, \
                descr_U, d_mVals, d_mRows, d_mCols, block_size, info_U, d_t, d_s, policy, d_buffer);
            // END TIME
            t_triSolve2 = second();
            t_triSolve_total += t_triSolve2 - t_triSolve1;

            // SPARE MATRIX VECTOR MULTIPLICATION PORTION OF ITERATION (second time around)
            // START TIME
            t_matVecMult1 = second();
            // spmv
            hipsparseDbsrmv(cusparseHandle, order, \
                operation, Nb, Nb, nnzb, &one, descr_M, \
                d_bVals, d_bRows, d_bCols, block_size, d_s, &zero, d_t);
            // END TIME
            t_matVecMult2 = second();
            t_matVecMult_total += t_matVecMult2 - t_matVecMult1;

            // apply wellContributions
            if(wellContribs.getNumWells() > 0){
                // START TIME
                t_wellContribs1 = second();
                wellContribs.apply(d_s, d_t);
                // END TIME
                t_wellContribs2 = second();
                t_wellContribs_total += t_wellcontribs2 - t_wellContribs1;
            }

            hipblasDdot(cublasHandle, n, d_t, 1, d_r, 1, &tmp1);
            hipblasDdot(cublasHandle, n, d_t, 1, d_t, 1, &tmp2);
            omega = tmp1 / tmp2;
            nomega = -omega;
            hipblasDaxpy(cublasHandle, n, &omega, d_s, 1, d_x, 1);
            hipblasDaxpy(cublasHandle, n, &nomega, d_t, 1, d_r, 1);

            hipblasDnrm2(cublasHandle, n, d_r, 1, &norm);


            if (norm < tolerance * norm_0 && it > minit) {
                break;
            }

            if (verbosity > 1) {
                std::ostringstream out;
                out << "it: " << it << std::scientific << ", norm: " << norm;
                OpmLog::info(out.str());
            }
        }

        t_total2 = second();

        res.iterations = std::min(it, (float)maxit);
        res.reduction = norm/norm_0;
        res.conv_rate  = static_cast<double>(pow(res.reduction,1.0/it));
        res.elapsed = t_total2 - t_total1;
        res.converged = (it != (maxit + 0.5));

        // TRANSFER TIMES FROM GPU TO CPU MEMORY (?)


        // copy the times and number of iterations to the csv file

        // open file for APPENDING
        std::ofstream myfile("/home/kenneth/work/rmine/opmTests/GPUTiming/gpu_linear_solve_time_details.csv", std::ios::app);
        // append to file
        myfile << it << "," << t_triSolve_total << "," << t_matVecMult_total << "," << t_wellContribs_total << "," << res.elapsed << "," << res.converged << "," << res.conv_rate <<"\n";
        myfile.close();
        // it , sparse tri solver time , sparse matrix vector multiplication time, wellContributions , total

        if (verbosity > 0) {
            std::ostringstream out;
            out << "=== converged: " << res.converged << ", conv_rate: " << res.conv_rate << ", time: " << res.elapsed << \
                   ", time per iteration: " << res.elapsed/it << ", iterations TEST: " << it; // added "TEST" to check if code is updated
            OpmLog::info(out.str());
        }
    }


    void cusparseSolverBackend::initialize(int N, int nnz, int dim) {
        this->N = N;
        this->nnz = nnz;
        this->block_size = dim;
        this->nnzb = nnz/block_size/block_size;
        Nb = (N + dim - 1) / dim;
        std::ostringstream out;
        out << "Initializing GPU, matrix size: " << N << " blocks, nnz: " << nnzb << " blocks";
        OpmLog::info(out.str());
        out.str("");
        out.clear();
        out << "Minit: " << minit << ", maxit: " << maxit << std::scientific << ", tolerance: " << tolerance;
        OpmLog::info(out.str());

        int deviceID = 0;
        hipSetDevice(deviceID);
        cudaCheckLastError("Could not get device");
        struct hipDeviceProp_t props;
        hipGetDeviceProperties(&props, deviceID);
        cudaCheckLastError("Could not get device properties");
        out.str("");
        out.clear();
        out << "Name GPU: " << props.name << ", Compute Capability: " << props.major << "." << props.minor;
        OpmLog::info(out.str());

        hipStreamCreate(&stream);
        cudaCheckLastError("Could not create stream");

        hipblasCreate(&cublasHandle);
        cudaCheckLastError("Could not create cublasHandle");

        hipsparseCreate(&cusparseHandle);
        cudaCheckLastError("Could not create cusparseHandle");

        hipMalloc((void**)&d_x, sizeof(double) * N);
        hipMalloc((void**)&d_b, sizeof(double) * N);
        hipMalloc((void**)&d_r, sizeof(double) * N);
        hipMalloc((void**)&d_rw,sizeof(double) * N);
        hipMalloc((void**)&d_p, sizeof(double) * N);
        hipMalloc((void**)&d_pw,sizeof(double) * N);
        hipMalloc((void**)&d_s, sizeof(double) * N);
        hipMalloc((void**)&d_t, sizeof(double) * N);
        hipMalloc((void**)&d_v, sizeof(double) * N);
        hipMalloc((void**)&d_bVals, sizeof(double) * nnz);
        hipMalloc((void**)&d_bCols, sizeof(double) * nnz);
        hipMalloc((void**)&d_bRows, sizeof(double) * (Nb+1));
        hipMalloc((void**)&d_mVals, sizeof(double) * nnz);
        cudaCheckLastError("Could not allocate enough memory on GPU");

        hipblasSetStream(cublasHandle, stream);
        cudaCheckLastError("Could not set stream to cublas");
        hipsparseSetStream(cusparseHandle, stream);
        cudaCheckLastError("Could not set stream to cusparse");

#if COPY_ROW_BY_ROW
        hipHostMalloc((void**)&vals_contiguous, sizeof(double) * nnz);
        cudaCheckLastError("Could not allocate pinned memory");
#endif

        initialized = true;
    } // end initialize()

    void cusparseSolverBackend::finalize() {
        if (initialized) {
            hipFree(d_x);
            hipFree(d_b);
            hipFree(d_r);
            hipFree(d_rw);
            hipFree(d_p);
            hipFree(d_pw);
            hipFree(d_s);
            hipFree(d_t);
            hipFree(d_v);
            hipFree(d_mVals);
            hipFree(d_bVals);
            hipFree(d_bCols);
            hipFree(d_bRows);
            hipFree(d_buffer);
            hipsparseDestroyBsrilu02Info(info_M);
            hipsparseDestroyBsrsv2Info(info_L);
            hipsparseDestroyBsrsv2Info(info_U);
            hipsparseDestroyMatDescr(descr_B);
            hipsparseDestroyMatDescr(descr_M);
            hipsparseDestroyMatDescr(descr_L);
            hipsparseDestroyMatDescr(descr_U);
            hipsparseDestroy(cusparseHandle);
            hipblasDestroy(cublasHandle);
#if COPY_ROW_BY_ROW
            hipHostFree(vals_contiguous);
#endif
            hipStreamDestroy(stream);
        }
    } // end finalize()


    void cusparseSolverBackend::copy_system_to_gpu(double *vals, int *rows, int *cols, double *b) {

        double t1, t2;
        if (verbosity > 2) {
            t1 = second();
        }

#if COPY_ROW_BY_ROW
        int sum = 0;
        for(int i = 0; i < Nb; ++i){
            int size_row = rows[i+1] - rows[i];
            memcpy(vals_contiguous + sum, vals + sum, size_row * sizeof(double) * block_size * block_size);
            sum += size_row * block_size * block_size;
        }
        hipMemcpyAsync(d_bVals, vals_contiguous, nnz * sizeof(double), hipMemcpyHostToDevice, stream);
#else
        hipMemcpyAsync(d_bVals, vals, nnz * sizeof(double), hipMemcpyHostToDevice, stream);
#endif

        hipMemcpyAsync(d_bCols, cols, nnz * sizeof(int), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(d_bRows, rows, (Nb+1) * sizeof(int), hipMemcpyHostToDevice, stream);
        hipMemcpyAsync(d_b, b, N * sizeof(double), hipMemcpyHostToDevice, stream);
        hipMemsetAsync(d_x, 0, sizeof(double) * N, stream);

        if (verbosity > 2) {
            hipStreamSynchronize(stream);
            t2 = second();
            std::ostringstream out;
            out << "cusparseSolver::copy_system_to_gpu(): " << t2-t1 << " s";
            OpmLog::info(out.str());
        }
    } // end copy_system_to_gpu()


    // don't copy rowpointers and colindices, they stay the same
    void cusparseSolverBackend::update_system_on_gpu(double *vals, int *rows, double *b) {

        double t1, t2;
        if (verbosity > 2) {
            t1 = second();
        }

#if COPY_ROW_BY_ROW
        int sum = 0;
        for(int i = 0; i < Nb; ++i){
            int size_row = rows[i+1] - rows[i];
            memcpy(vals_contiguous + sum, vals + sum, size_row * sizeof(double) * block_size * block_size);
            sum += size_row * block_size * block_size;
        }
        hipMemcpyAsync(d_bVals, vals_contiguous, nnz * sizeof(double), hipMemcpyHostToDevice, stream);
#else
        hipMemcpyAsync(d_bVals, vals, nnz * sizeof(double), hipMemcpyHostToDevice, stream);
#endif

        hipMemcpyAsync(d_b, b, N * sizeof(double), hipMemcpyHostToDevice, stream);
        hipMemsetAsync(d_x, 0, sizeof(double) * N, stream);

        if (verbosity > 2) {
            hipStreamSynchronize(stream);
            t2 = second();
            std::ostringstream out;
            out << "cusparseSolver::update_system_on_gpu(): " << t2-t1 << " s";
            OpmLog::info(out.str());
        }
    } // end update_system_on_gpu()


    void cusparseSolverBackend::reset_prec_on_gpu() {
        hipMemcpyAsync(d_mVals, d_bVals, nnz  * sizeof(double), hipMemcpyDeviceToDevice, stream);
    }


    bool cusparseSolverBackend::analyse_matrix() {

        int d_bufferSize_M, d_bufferSize_L, d_bufferSize_U, d_bufferSize;
        double t1, t2;

        if (verbosity > 2) {
            t1 = second();
        }

        hipsparseCreateMatDescr(&descr_B);
        hipsparseCreateMatDescr(&descr_M);
        hipsparseSetMatType(descr_B, HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatType(descr_M, HIPSPARSE_MATRIX_TYPE_GENERAL);
        const hipsparseIndexBase_t base_type = HIPSPARSE_INDEX_BASE_ZERO;     // matrices from Flow are base0

        hipsparseSetMatIndexBase(descr_B, base_type);
        hipsparseSetMatIndexBase(descr_M, base_type);

        hipsparseCreateMatDescr(&descr_L);
        hipsparseSetMatIndexBase(descr_L, base_type);
        hipsparseSetMatType(descr_L, HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatFillMode(descr_L, HIPSPARSE_FILL_MODE_LOWER);
        hipsparseSetMatDiagType(descr_L, HIPSPARSE_DIAG_TYPE_UNIT);

        hipsparseCreateMatDescr(&descr_U);
        hipsparseSetMatIndexBase(descr_U, base_type);
        hipsparseSetMatType(descr_U, HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatFillMode(descr_U, HIPSPARSE_FILL_MODE_UPPER);
        hipsparseSetMatDiagType(descr_U, HIPSPARSE_DIAG_TYPE_NON_UNIT);
        cudaCheckLastError("Could not initialize matrix descriptions");

        hipsparseCreateBsrilu02Info(&info_M);
        hipsparseCreateBsrsv2Info(&info_L);
        hipsparseCreateBsrsv2Info(&info_U);
        cudaCheckLastError("Could not create analysis info");

        hipsparseDbsrilu02_bufferSize(cusparseHandle, order, Nb, nnzb,
            descr_M, d_bVals, d_bRows, d_bCols, block_size, info_M, &d_bufferSize_M);
        hipsparseDbsrsv2_bufferSize(cusparseHandle, order, operation, Nb, nnzb,
            descr_L, d_bVals, d_bRows, d_bCols, block_size, info_L, &d_bufferSize_L);
        hipsparseDbsrsv2_bufferSize(cusparseHandle, order, operation, Nb, nnzb,
            descr_U, d_bVals, d_bRows, d_bCols, block_size, info_U, &d_bufferSize_U);
        cudaCheckLastError();
        d_bufferSize = std::max(d_bufferSize_M, std::max(d_bufferSize_L, d_bufferSize_U));
        
        hipMalloc((void**)&d_buffer, d_bufferSize);

        // analysis of ilu LU decomposition
        hipsparseDbsrilu02_analysis(cusparseHandle, order, \
            Nb, nnzb, descr_B, d_bVals, d_bRows, d_bCols, \
            block_size, info_M, policy, d_buffer);

        int structural_zero;
        hipsparseStatus_t status = hipsparseXbsrilu02_zeroPivot(cusparseHandle, info_M, &structural_zero);
        if (HIPSPARSE_STATUS_ZERO_PIVOT == status) {
            return false;
        }

        // analysis of ilu apply
        hipsparseDbsrsv2_analysis(cusparseHandle, order, operation, \
            Nb, nnzb, descr_L, d_bVals, d_bRows, d_bCols, \
            block_size, info_L, policy, d_buffer);

        hipsparseDbsrsv2_analysis(cusparseHandle, order, operation, \
            Nb, nnzb, descr_U, d_bVals, d_bRows, d_bCols, \
            block_size, info_U, policy, d_buffer);
        cudaCheckLastError("Could not analyse level information");

        if (verbosity > 2) {
            hipStreamSynchronize(stream);
            t2 = second();
            std::ostringstream out;
            out << "cusparseSolver::analyse_matrix(): " << t2-t1 << " s";
            OpmLog::info(out.str());
        }

        analysis_done = true;

        return true;
    } // end analyse_matrix()

    bool cusparseSolverBackend::create_preconditioner() {

        double t1, t2;
        if (verbosity > 2) {
            t1 = second();
        }

        d_mCols = d_bCols;
        d_mRows = d_bRows;
        hipsparseDbsrilu02(cusparseHandle, order, \
            Nb, nnzb, descr_M, d_mVals, d_mRows, d_mCols, \
            block_size, info_M, policy, d_buffer);
        cudaCheckLastError("Could not perform ilu decomposition");

        int structural_zero;
        // hipsparseXbsrilu02_zeroPivot() calls hipDeviceSynchronize()
        hipsparseStatus_t status = hipsparseXbsrilu02_zeroPivot(cusparseHandle, info_M, &structural_zero);
        if (HIPSPARSE_STATUS_ZERO_PIVOT == status) {
            return false;
        }

        if (verbosity > 2) {
            hipStreamSynchronize(stream);
            t2 = second();
            std::ostringstream out;
            out << "cusparseSolver::create_preconditioner(): " << t2-t1 << " s" << "TESTING";
            OpmLog::info(out.str());
        }
        return true;
    } // end create_preconditioner()


    void cusparseSolverBackend::solve_system(WellContributions& wellContribs, BdaResult &res) {
        // actually solve
        gpu_pbicgstab(wellContribs, res);
        hipStreamSynchronize(stream);
        cudaCheckLastError("Something went wrong during the GPU solve");
    } // end solve_system()


    // copy result to host memory
    // caller must be sure that x is a valid array
    void cusparseSolverBackend::post_process(double *x) {

        double t1, t2;
        if (verbosity > 2) {
            t1 = second();
        }

        hipMemcpyAsync(x, d_x, N * sizeof(double), hipMemcpyDeviceToHost, stream);
        hipStreamSynchronize(stream);

        if (verbosity > 2) {
            t2 = second();
            std::ostringstream out;
            out << "cusparseSolver::post_process(): " << t2-t1 << " s";
            OpmLog::info(out.str());
        }
    } // end post_process()


    typedef cusparseSolverBackend::cusparseSolverStatus cusparseSolverStatus;

    cusparseSolverStatus cusparseSolverBackend::solve_system(int N, int nnz, int dim, double *vals, int *rows, int *cols, double *b, WellContributions& wellContribs, BdaResult &res) { 
        if (initialized == false) {
            initialize(N, nnz, dim);
            copy_system_to_gpu(vals, rows, cols, b);
        }else{
            update_system_on_gpu(vals, rows, b);
        }
        if (analysis_done == false) {
            if (!analyse_matrix()) {
                return cusparseSolverStatus::CUSPARSE_SOLVER_ANALYSIS_FAILED;
            }
        }
        reset_prec_on_gpu();
        if (create_preconditioner()) {
            solve_system(wellContribs, res);
        }else{
            return cusparseSolverStatus::CUSPARSE_SOLVER_CREATE_PRECONDITIONER_FAILED;
        }
        return cusparseSolverStatus::CUSPARSE_SOLVER_SUCCESS;
    }


}


